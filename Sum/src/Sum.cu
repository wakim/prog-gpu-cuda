/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, "Error %s at line %d in file %s\n",				\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);	\
		exit(1);														\
	}																	\
}

float scan_seq(float *, int, int);
int calculate_grid_size(int, int);
float scan(float*, int, int, char*, int);
void fprint_mat(FILE *, float *, int);
float* init(int);

__global__ void scan_kernel(float *, int);
__global__ void scan_kernel_coalesc(float *, int);

int block_size;

int main(int argc, char **argv) {
	if(argc < 4) {
		printf("usage: %s <tamanho_vetor> <algoritmo> <tamanho_bloco> <debug opcional>\n\
algoritmo:\n\ts (Sequencial)\n\tc (Cuda nao coalescente)\n\tcc (Cuda coalescente)\n", argv[0]);
		exit(-1);
	}

	FILE *input = fopen(argv[1], "r");

	int debug = (argc > 4 ? strcmp("debug", argv[4]) == 0 : 0);
	int size = atoi(argv[1]);
	float *vector = init(size);
	float size_of = size * sizeof(float);
	float mega_byte = 1024 * 1024;

	block_size = atoi(argv[3]);

	printf("%f MBytes\n",  (size_of / mega_byte));

	if(strncmp(argv[2], "s", 1) == 0) {
		float sum = scan_seq(vector, size, debug);
		printf("Sum: %f\n", sum);
	} else {
		float sum = scan(vector, block_size, size, argv[2], debug);
		printf("Sum: %f\n", sum);
	}

	return 0;
}

int calculate_grid_size(int tile_size, int size) {
	int div = size / tile_size;
	int remainder = size - (tile_size * div);

	if(remainder > 0) {
		div++;
	}

	return div;
}

float scan(float *input, int tile_size, int size, char *algorithm, int debug) {
	int dim_grid = calculate_grid_size(tile_size, size);
	float *device_input;
	FILE *f = NULL;

	size_t size_bytes = size * sizeof(float);
	size_t block_size_bytes = tile_size * sizeof(float);

	CUDA_CHECK_RETURN(hipMalloc((void **) &device_input, size_bytes));
	CUDA_CHECK_RETURN(hipMemcpy(device_input, input, size_bytes, hipMemcpyHostToDevice));

	if(strlen(algorithm) == 1 && strncmp(algorithm, "c", 1) == 0) {
		scan_kernel <<<dim_grid, tile_size>>> (device_input, size);
		if(debug) {
			f = fopen("saida_c.txt", "w");
		}
	} else {
		scan_kernel_coalesc <<<dim_grid, tile_size, block_size_bytes>>> (device_input, size);
		if(debug) {
			f = fopen("saida_cc.txt", "w");
		}
	}

	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(input, device_input, size_bytes, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(device_input));

	// Percorro as bordas dos blocos para pegar o menor
	int index = 0;
	float sum = 0;
	float sum1 = 0.0, sum2 = 0.0;
	int half_dim_grid = dim_grid / 2;

	if(debug) {
		fprint_mat(f, input, size);
		fclose(f);
	}

	// Soma as bordas, soma a metade para evitar de perder precisao
	for(int i = 1; i < half_dim_grid; ++i) {
		index = (tile_size * i) - 1;
		sum1 += input[index];
	}

	for(int i = half_dim_grid; i < dim_grid; ++i) {
		index = (tile_size * i) - 1;
		sum2 += input[index];
	}

	sum = sum1;

	if((size) != index) {
		sum += input[size - 1];
	}

	sum += sum2;

	return sum;
}

float scan_seq(float *input, int size, int debug) {
	float found = 0;

	for(int i = 0; i < size; ++i) {
		found += input[i];
	}

	if(debug) {
		FILE *f = fopen("saida_s.txt", "w");
		fprint_mat(f, input, size);
		fclose(f);
	}

	return found;
}

void fprint_mat(FILE *f, float *v, int n) {
	for(int i = 0; i < n-1; i++) {
		fprintf(f, "%f ", v[i]);
	}

	fprintf(f, "%f", v[n-1]);
}

float* init(int n) {
	float *v = (float*) malloc(sizeof(float) * n);

	for(int i = 0; i < n; ++i) {
		v[i] = (float) i;
	}

	return v;
}

////////////////////////////////////////////////////////////////////////////////

/***
 * Esse Scan é feito apenas no interior do bloco, logo quem chamou ele precisa verificar
 * os limites dos blocos para pegar o menor valor
 */
__global__
void scan_kernel(float *input, int size) {
	int local_block_dim = blockDim.x;
	int bi = local_block_dim * blockIdx.x;
	int ti = bi + threadIdx.x;
	int ti_bi = ti - bi; // ti - bi evita que a thread saia do limite do bloco.
	int local_size = size;
	float aux = 0;
	int offset = 1;

	// Algoritmo do Scan em cima das posições e threads relativas ao bloco
	for(offset = 1; offset < local_block_dim; offset *= 2) {

		if(ti_bi >= offset && ti < local_size) {
			aux = input[ti - offset];
		}

		__syncthreads();

		if(ti_bi >= offset && ti < local_size) {
			input[ti] = aux + input[ti];
		}

		__syncthreads();
	}
}

__global__
void scan_kernel_coalesc(float *input, int size) {
	int local_block_dim = blockDim.x;
	int thread_id = threadIdx.x;
	int bi = local_block_dim * blockIdx.x;
	int ti = bi + threadIdx.x;
	int local_size = size;
	float aux = 0;
	int offset = 1;

	extern __shared__ float local[];

	// Carrega o valor relativo a thread para a memória local.
	if(ti < local_size) {
		local[thread_id] = input[ti];
	}

	__syncthreads();

	// Aplica o Scan em cima da memória local
	for(offset = 1; offset < local_block_dim; offset *= 2) {

		if(thread_id >= offset && thread_id < local_block_dim) {
			aux = local[thread_id - offset];
		}

		__syncthreads();

		if(thread_id >= offset && thread_id < local_block_dim) {
			local[thread_id] = aux + local[thread_id];
		}

		__syncthreads();
	}

	// Volta com o valor local na memória global
	input[ti] = local[thread_id];
}
