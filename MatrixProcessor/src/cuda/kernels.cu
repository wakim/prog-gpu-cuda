#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <stdio.h>
#include <typeinfo>

#include "kernels.cu.h"
#include "reduce_kernels.cu"
#include "reduce_kernels_specialized.cu"
#include "general_kernels.cu"
#include "general_kernels_specialized.cu"

template <typename T, int max_threads>
T* transpose_matrix(T* matrix_device, dim3 block, dim3 grid, int rows, int cols) {
	T* t_matrix_device = NULL;
	int shared_array_size_bytes = block.x * (block.y + 1) * sizeof(T);

	CudaUtils::malloc_on_gpu<T, false>((void**) &t_matrix_device, cols, rows);

	transpose<T> <<<grid, block, shared_array_size_bytes>>>(t_matrix_device, matrix_device, cols, rows);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	return t_matrix_device;
}

template <typename T, typename Operation, typename SharedMatrixSetter, bool specialized>
T* apply_reduce_operation_line(dim3& block, dim3& grid, T* matrix_device,
	int rows, int cols, Operation op, SharedMatrixSetter setter) {

	return apply_reduce_operation_line<T, Operation, SharedMatrixSetter, SharedMatrixSetter, specialized>
	(
		block, grid, matrix_device, rows, cols, op, setter, setter
	);
}

/**
 * Aplica uma determinada operação em cada linha da matriz em paralelo
 */
template <typename T, typename Operation, typename SharedMatrixSetter1, typename SharedMatrixSetter2, bool specialized>
T* apply_reduce_operation_line(dim3& block, dim3& grid, T* matrix_device,
	int rows, int cols, Operation op, SharedMatrixSetter1 setter1, SharedMatrixSetter2 setter2) {
	T* block_matrix_device = NULL;
	int shared_matrix_size = block.x * block.y;
	int shared_matrix_size_bytes = shared_matrix_size * sizeof(T);

	int block_matrix_cols = grid.x;

	CudaUtils::malloc_on_gpu<T, false>((void**) &block_matrix_device, rows, block_matrix_cols);

	//printf("\n\n\nmatrix_device on reduce %s %s %s %s\n\n\n", typeid(T).name(),
	//	typeid(Operation).name(), typeid(SharedMatrixSetter1).name(), typeid(SharedMatrixSetter2).name()
	//);

	//CudaUtils::load_and_print<T, true>(matrix_device, rows, cols);

	if(specialized) {
		if(typeid(Operation) == typeid(MaxReduceOperation<T>) && typeid(SharedMatrixSetter1) == typeid(DefaultSharedPositionSetter<T>)) {
			multiple_reduce_max_default_setter<T> <<<grid, block, shared_matrix_size_bytes>>>
				(matrix_device, block_matrix_device, rows, cols);
		} else if(typeid(Operation) == typeid(SumReduceOperation<T>)) {
			multiple_reduce_sum_dfi_setter<T> <<<grid, block, shared_matrix_size_bytes>>>
				(matrix_device, block_matrix_device, rows, cols);
		}
	} else {
		multiple_reduce<T> <<<grid, block, shared_matrix_size_bytes>>>
			(matrix_device, block_matrix_device, rows, cols, op, setter1);
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	//printf("\n\n\nblock_matrix_device on reduce %s %d %d %d %d\n\n\n", typeid(T).name(), rows,
	//	block_matrix_cols, grid.x, grid.y
	//);

	//CudaUtils::load_and_print<T, true>(block_matrix_device, rows, block_matrix_cols);

	block.y = 1;
	block.x = block_matrix_cols;
	grid.y = 1;
	grid.x = rows;

	shared_matrix_size = block.x;
	shared_matrix_size_bytes = block.x * sizeof(T);

	if(specialized) {
		if(typeid(Operation) == typeid(MaxReduceOperation<T>) && typeid(SharedMatrixSetter2) == typeid(DefaultSharedPositionSetter<T>)) {
			multiple_reduce_2_max_default_setter<T> <<<grid, block, shared_matrix_size_bytes>>>
			(
				block_matrix_device, rows, block_matrix_cols
			);
		} else if(typeid(Operation) == typeid(SumReduceOperation<T>)) {
			multiple_reduce_2_sum_default_setter<T> <<<grid, block, shared_matrix_size_bytes>>>
			(
				block_matrix_device, rows, block_matrix_cols
			);
		}
	} else {
		multiple_reduce_2<T> <<<grid, block, shared_matrix_size_bytes>>>
		(
			block_matrix_device, rows, block_matrix_cols, op, setter2
		);
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	//printf("\n\n\nblock_matrix_device on reduce2 %s %d %d\n\n\n", typeid(T).name(), rows, block_matrix_cols);
	//CudaUtils::load_and_print<T, true>(block_matrix_device, rows, block_matrix_cols);

	return block_matrix_device;
}

/**
 * Normaliza a matriz linha a linha
 */
template <typename T, typename N, int max_threads, bool specialized>
N* normalize_matrix_kernel_wrapper(dim3 matrix_block, dim3 matrix_grid,
	T* matrix_device, int rows, int cols) {

	dim3 max_matrix_block(matrix_block), max_matrix_grid(matrix_grid);

	//printf("\n\nmatrix_device\n\n");
	//CudaUtils::load_and_print<T, false>(matrix_device, rows, cols, rows - 10, cols - 10);

	T* max_matrix_device = apply_reduce_operation_line<T, MaxReduceOperation<T>, DefaultSharedPositionSetter<T>, specialized>
	(
		max_matrix_block, max_matrix_grid, matrix_device, rows, cols,
		MaxReduceOperation<T>(), DefaultSharedPositionSetter<T>()
	);

	//printf("\nNORMALIZE %d %d %s\n", rows, cols, typeid(T).name());
	//printf("\n\nmax_matrix_device\n\n");
	//CudaUtils::load_and_print<T, false>(max_matrix_device, rows, max_matrix_block.x);

	dim3 t_max_matrix_block, t_max_matrix_grid;
	CudaUtils::compute_num_threads_blocks(t_max_matrix_grid, t_max_matrix_block, max_threads, rows, max_matrix_block.x, true, true);
	T* t_max_matrix_device = transpose_matrix<T, max_threads>(max_matrix_device, t_max_matrix_block, t_max_matrix_grid, rows, max_matrix_block.x);
	
	T* t_max_matrix_device_last_line = & t_max_matrix_device[(max_matrix_block.x - 1) * (rows)];

	//printf("\n\nt_max_matrix_device\n\n");
	//CudaUtils::load_and_print<T, true>(t_max_matrix_device, max_matrix_block.x, rows);
	//printf("\n\n_t_max_matrix_device_last_line\n\n");
	//CudaUtils::load_and_print<T, true>(t_max_matrix_device_last_line, 1, rows);

	CudaUtils::compute_num_threads_blocks(t_max_matrix_grid, t_max_matrix_block, max_threads, 1, rows, false, true);

	T* t_max_matrix_reduced = apply_reduce_operation_line<T, MaxReduceOperation<T>, DefaultSharedPositionSetter<T>, specialized>
	(
		t_max_matrix_block, t_max_matrix_grid, t_max_matrix_device_last_line, 1, rows,
		MaxReduceOperation<T>(), DefaultSharedPositionSetter<T>()
	);

	//printf("\n\n_t_max_matrix_device_last_line_reduced\n\n");
	//CudaUtils::load_and_print<T, true>(t_max_matrix_reduced, 1, t_max_matrix_block.x, 0, 0);

	N* matrix_result_device = NULL;
	CudaUtils::malloc_on_gpu<T, false>((void**) &matrix_result_device, rows, cols);

	//printf("t_m_m_b.x %d rows %d cols %d\n", t_max_matrix_block.x, rows, cols);
	//printf("b_x %d b_y %d g_x %d g_y %d\n", matrix_block.x, matrix_block.y, matrix_grid.x, matrix_grid.y);
	//printf("VAI CHAMAR O NORMALIZE\n");

	if(typeid(T) == typeid(N)) {
		if(specialized) {
			multiple_normalize_non_cast<T, N> <<<matrix_grid, matrix_block>>>
			(
				matrix_result_device, matrix_device, t_max_matrix_reduced,
				rows, cols, t_max_matrix_block.x
			);
		} else {
			multiple_normalize<T, N> <<<matrix_grid, matrix_block>>>
			(
				matrix_result_device, matrix_device, t_max_matrix_reduced,
				rows, cols, t_max_matrix_block.x, NonCastNormalizeMatrixOperation<T, N>()
			);
		}
	} else {
		if(specialized) {
			multiple_normalize_cast<T, N> <<<matrix_grid, matrix_block>>>
			(
				matrix_result_device, matrix_device, t_max_matrix_reduced,
				rows, cols, t_max_matrix_block.x
			);
		} else {
			multiple_normalize<T, N> <<<matrix_grid, matrix_block>>>
			(
				matrix_result_device, matrix_device, t_max_matrix_reduced,
				rows, cols, t_max_matrix_block.x, CastNormalizeMatrixOperation<T, N>()
			);
		}
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipFree(max_matrix_device));
	CUDA_CHECK_RETURN(hipFree(matrix_device));
	CUDA_CHECK_RETURN(hipFree(t_max_matrix_device));
	CUDA_CHECK_RETURN(hipFree(t_max_matrix_reduced));

	//printf("\n\nresult_matrix_device\n\n");
	//CudaUtils::load_and_print<T, false>(matrix_result_device, rows, cols);
	//CudaUtils::load_and_print<T, false>(matrix_result_device, rows, cols, rows - 10, cols - 10);

	return matrix_result_device;
}

template <typename T, typename N, int max_threads, bool specialized>
N* apply_tf_idf_metrics(dim3 matrix_block, dim3 matrix_grid, T* matrix_device, int rows, int cols) {

	dim3 dfi_matrix_block(matrix_block), dfi_matrix_grid(matrix_grid);
	dim3 max_fj_matrix_block(matrix_block), max_fj_matrix_grid(matrix_grid);
	dim3 matrix_transpose_block(matrix_block), matrix_transpose_grid(matrix_grid);

	CudaUtils::compute_num_threads_blocks(matrix_transpose_grid, matrix_transpose_block, max_threads, rows, cols, true, false);
	CudaUtils::compute_num_threads_blocks(max_fj_matrix_grid, max_fj_matrix_block, max_threads, cols, rows, false, false);

	N* tf_idf_matrix_device = NULL;
	T* transposed_matrix_device = NULL;
	T* max_fj_matrix_device = NULL;

	//printf("TF_IDF (DFI_MAX)\n\n");
	// Para cada linha da matriz, o valor do dfi está na última coluna.
	T* dfi_matrix_device = apply_reduce_operation_line<T, SumReduceOperation<T>, DFISharedPositionSetter<T>, DefaultSharedPositionSetter<T>, specialized>
	(
		dfi_matrix_block, dfi_matrix_grid, matrix_device, rows, cols,
		SumReduceOperation<T>(), DFISharedPositionSetter<T>(), DefaultSharedPositionSetter<T>()
	);

	//printf("\n\n(dfi_matrix_device)\n\n");
	//CudaUtils::load_and_print<T, true>(dfi_matrix_device, rows, dfi_matrix_block.x);

	transposed_matrix_device = transpose_matrix<T, max_threads>(matrix_device, matrix_transpose_block, matrix_transpose_grid, rows, cols);

	//CudaUtils::load_and_print<int, true>(transposed_matrix_device, cols, rows, "%d ");

	// Para cada linha da matriz, o valor do max(fj) está na última coluna.
	// O número da coluna na matrix TxD indexa uma linha na matriz do max(fj)
	//printf("TF_IDF (MAX_FJ)\n\n");
	max_fj_matrix_device = apply_reduce_operation_line<T, MaxReduceOperation<T>, DefaultSharedPositionSetter<T>, specialized>
	(
		max_fj_matrix_block, max_fj_matrix_grid, transposed_matrix_device,
		cols, rows, MaxReduceOperation<T>(), DefaultSharedPositionSetter<T>()
	);

	CUDA_CHECK_RETURN(hipFree(transposed_matrix_device));

	//printf("\n\n(max_fj_matrix)\n\n");
	//CudaUtils::load_and_print<T, true>(max_fj_matrix_device, cols, max_fj_matrix_block.x);
	
	CudaUtils::malloc_on_gpu<N, false>((void**) &tf_idf_matrix_device, rows, cols);
	
	if(typeid(T) == typeid(N)) {
		if(specialized) {
			transform_tf_idf_non_cast<T, N> <<<matrix_grid, matrix_block>>>
			(
				tf_idf_matrix_device, matrix_device, rows, cols,
				dfi_matrix_device, max_fj_matrix_device,
				dfi_matrix_block.x, max_fj_matrix_block.x
			);
		} else {
			transform_tf_idf<T, N> <<<matrix_grid, matrix_block>>>
			(
				tf_idf_matrix_device, matrix_device, rows, cols,
				dfi_matrix_device, max_fj_matrix_device,
				dfi_matrix_block.x, max_fj_matrix_block.x,
				NonCastTFIDFOperation<T, N>()
			);
		}
	} else {
		if(specialized) {
			transform_tf_idf_cast<T, N> <<<matrix_grid, matrix_block>>>
			(
				tf_idf_matrix_device, matrix_device, rows, cols,
				dfi_matrix_device, max_fj_matrix_device,
				dfi_matrix_block.x, max_fj_matrix_block.x
			);
		} else {
			transform_tf_idf<T, N> <<<matrix_grid, matrix_block>>>
			(
				tf_idf_matrix_device, matrix_device, rows, cols,
				dfi_matrix_device, max_fj_matrix_device,
				dfi_matrix_block.x, max_fj_matrix_block.x,
				CastTFIDFOperation<T, N>()
			);
		}
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	//CudaUtils::load_and_print<N, true>(tf_idf_matrix_device, rows, cols, rows - 10, cols - 10);

	CUDA_CHECK_RETURN(hipFree(matrix_device));
	CUDA_CHECK_RETURN(hipFree(dfi_matrix_device));
	CUDA_CHECK_RETURN(hipFree(max_fj_matrix_device));

	return tf_idf_matrix_device;
}