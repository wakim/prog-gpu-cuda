#include "hip/hip_runtime.h"

#include "shared_memory.cu.h"

/**
 * Aplica o algoritmo redu��o no vetor dado.
**/
template <typename T, typename ReduceOperation>
__device__ void reduce(T* shared_array, int array_limit,
	int ti, int shared_pos, bool in_bounds, ReduceOperation op) {
	int offset = 1;
	T local_aux = static_cast<T>(-1);

	for(offset = 1; offset < array_limit; offset <<= 1) {

		if(in_bounds && ti >= offset) {
			local_aux = shared_array[shared_pos - offset];
		}

		__syncthreads();

		if(in_bounds && ti >= offset) {
			op(local_aux, shared_array, shared_pos);
		}

		__syncthreads();
	}
}

/**
 * Nesse kernel, cada grid � uma linha da matrix dos resultados parciais do kernel anterior
 * E s� � lan�ado um bloco por linha
 */
template <typename T, typename ReduceOperation, typename SharedPositionSetter>
__global__ void multiple_reduce_2(T* block_matrix, int rows, int cols,
	ReduceOperation op, SharedPositionSetter setter) {

	unsigned int ti = threadIdx.x, bx = blockIdx.x;
	unsigned int matrix_pos = bx * cols + ti;
	bool in_bounds = bx < rows && ti < cols;

	SharedMemory<T> shared_memory;
	T* shared_array = shared_memory.getPointer();

	if(in_bounds) {
		setter(shared_array, ti, block_matrix, matrix_pos);
	}

	__syncthreads();

	reduce(shared_array, cols, ti, ti, in_bounds, op);

	if(in_bounds) {
		block_matrix[matrix_pos] = shared_array[ti];
	}
}

/*
 * Esse kernel far� a redu��o modificada em cada linha da matriz, para obter o maior valor dela.
 * Esse kernel s� faz a redu��o intra bloco, um segundo kernel precisa ser chamado para finalizar
 * o calculo.
 * @matrix � a matriz inteira
 * @block_matrix � uma matriz temporaria para o resultado da redu��o de cada bloco
 * block_matrix ser� usada depois no multiple_max2.
 */
template <typename T, typename ReduceOperation, typename SharedMatrixSetter>
__global__ void multiple_reduce(T* matrix, T* block_matrix, int rows, int cols,
	ReduceOperation op, SharedMatrixSetter setter) {

	unsigned int tx = threadIdx.x, ty = threadIdx.y;
	unsigned int bx = blockIdx.x, by = blockIdx.y;
	unsigned int bdx = blockDim.x, bdy = blockDim.y;
	unsigned int col = (bx * bdx) + tx;
	unsigned int row = (by * bdy) + ty;
	unsigned int matrix_pos = row * cols + col, shared_pos = (ty * bdx + tx);
	bool in_bounds = row < rows && col < cols;

	// � do tamanho do bloco
	SharedMemory<T> shared_memory;
	T* shared_array = shared_memory.getPointer();

	if(in_bounds) {
		setter(shared_array, shared_pos, matrix, matrix_pos);
		//if(tx == 0 && ty == 0 &&bx == 0 && by == 0) {
		//	printf("tx %u ty %u value %f\n", tx, ty, shared_array[shared_pos]);
		//}
	}

	__syncthreads();

	reduce(shared_array, bdx, tx, shared_pos, in_bounds, op);

	if(in_bounds && (tx == (bdx - 1) || col == (cols - 1))) {
		unsigned int v = row * gridDim.x + bx;

		block_matrix[v] = shared_array[shared_pos];

		//int base_pos = ty * bdx;

		//if(row == 0) {
		//	for(int i = 0; i < bdx; ++i) {
		//		printf("pos %d (%d: %f)\n", matrix_pos, i, shared_array[base_pos + i]);
		//	}
		//}

		//if(ty == 0 && ) {
		//	printf("tx %u ty %u value %f\n", tx, ty, shared_array[shared_pos]);
		//}
	}
}