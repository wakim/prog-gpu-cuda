#include "hip/hip_runtime.h"

#include "shared_memory.cu.h"

template <typename T, typename N>
__global__ void multiple_normalize_cast(N* output_matrix, T* input_matrix, T* max_block_matrix, int rows, int cols,
	int block_matrix_cols) {

	unsigned int tx = threadIdx.x, ty = threadIdx.y;
	unsigned int bx = blockIdx.x, by = blockIdx.y;
	unsigned int bdx = blockDim.x, bdy = blockDim.y;
	unsigned int col = (bx * bdx) + tx;
	unsigned int row = (by * bdy) + ty;
	unsigned int matrix_pos = row * cols + col;
	bool in_bounds = row < rows && col < cols;
	
	if(in_bounds) {
		unsigned int block_matrix_pos = block_matrix_cols - 1; // Pega o �ltimo elemento
		
		T value = max_block_matrix[block_matrix_pos];

		if(value != static_cast<T>(0)) {
			output_matrix[matrix_pos] = (static_cast<N>(input_matrix[matrix_pos]) / static_cast<N>(max_block_matrix[block_matrix_pos]));
		} else {
			output_matrix[matrix_pos] = static_cast<N>(0);
		}
	}
}

template <typename T, typename N>
__global__ void multiple_normalize_non_cast(N* output_matrix, T* input_matrix, T* max_block_matrix, int rows, int cols,
	int block_matrix_cols) {

	unsigned int tx = threadIdx.x, ty = threadIdx.y;
	unsigned int bx = blockIdx.x, by = blockIdx.y;
	unsigned int bdx = blockDim.x, bdy = blockDim.y;
	unsigned int col = (bx * bdx) + tx;
	unsigned int row = (by * bdy) + ty;
	unsigned int matrix_pos = row * cols + col;
	bool in_bounds = row < rows && col < cols;
	
	if(in_bounds) {
		unsigned int block_matrix_pos = block_matrix_cols - 1; // Pega o �ltimo elemento
		
		T value = max_block_matrix[block_matrix_pos];

		if(value != static_cast<T>(0)) {
			output_matrix[matrix_pos] = (input_matrix[matrix_pos] / max_block_matrix[block_matrix_pos]);
		} else {
			output_matrix[matrix_pos] = static_cast<N>(0);
		}
	}
}

template <typename T, typename N>
__global__ void transform_tf_idf_cast(N* output_matrix, T* input_matrix, int rows, int cols,
	T* dfi_matrix, T* max_fj_matrix, int dfi_matrix_cols, int max_fj_matrix_cols) {

	unsigned int tx = threadIdx.x, ty = threadIdx.y;
	unsigned int bx = blockIdx.x, by = blockIdx.y;
	unsigned int bdx = blockDim.x, bdy = blockDim.y;
	unsigned int col = (bx * bdx) + tx;
	unsigned int row = (by * bdy) + ty;
	unsigned int matrix_pos = row * cols + col;
	bool in_bounds = row < rows && col < cols;

	if(in_bounds) {
		unsigned int dfi_matrix_pos = (row * dfi_matrix_cols) + (dfi_matrix_cols - 1);
		unsigned int max_fj_matrix_pos = (col * max_fj_matrix_cols) + (max_fj_matrix_cols - 1);
		
		T max_fj = max_fj_matrix[max_fj_matrix_pos];
		T dfi = dfi_matrix[dfi_matrix_pos];
		
		N tf_ij = max_fj == 0 ? 0 : static_cast<N>(input_matrix[matrix_pos]) / static_cast<N>(max_fj);
		N idf_i = dfi == 0 ? 0 : log10(static_cast<N>(cols) / static_cast<N>(dfi));

		output_matrix[matrix_pos] = tf_ij * idf_i;
	}
}

template <typename T, typename N>
__global__ void transform_tf_idf_non_cast(N* output_matrix, T* input_matrix, int rows, int cols,
	T* dfi_matrix, T* max_fj_matrix, int dfi_matrix_cols, int max_fj_matrix_cols) {

	unsigned int tx = threadIdx.x, ty = threadIdx.y;
	unsigned int bx = blockIdx.x, by = blockIdx.y;
	unsigned int bdx = blockDim.x, bdy = blockDim.y;
	unsigned int col = (bx * bdx) + tx;
	unsigned int row = (by * bdy) + ty;
	unsigned int matrix_pos = row * cols + col;
	bool in_bounds = row < rows && col < cols;

	if(in_bounds) {
		unsigned int dfi_matrix_pos = (row * dfi_matrix_cols) + (dfi_matrix_cols - 1);
		unsigned int max_fj_matrix_pos = (col * max_fj_matrix_cols) + (max_fj_matrix_cols - 1);
		
		T max_fj = max_fj_matrix[max_fj_matrix_pos];
		T dfi = dfi_matrix[dfi_matrix_pos];
		
		N tf_ij = max_fj == 0 ? 0 : input_matrix[matrix_pos] / max_fj;
		N idf_i = dfi == 0 ? 0 : log10(static_cast<N>(cols) / dfi);

		output_matrix[matrix_pos] = tf_ij * idf_i;
	}
}