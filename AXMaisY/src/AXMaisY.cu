/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

void alloc_on_device(float *, float*, void **, void **, int);
void free_on_device(void *, void*);
void copy_from_device(void *, float*, int);
void fprint_mat(FILE *, float *, int);
void a_x_mais_y_host(int, float *, float*, int);
float* init(int);

__global__ void a_x_mais_y_device(int, float* , float*, int);
__global__ void a_x_mais_y_device_coalesce(int, float*, float*, int);

const int SEQUENTIAL = 1;
const int CUDA = 2;
const int CUDA_COALESCE = 4;

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char ** argv) {

	if(argc < 4) {
		printf("Uso: %s <algoritmo> <tamanho_vetor> <tamanho_bloco> <a opcional> <debug opcional>\n\
algoritmo:\n\ts (Sequencial)\n\tc (Cuda nao coalescente)\n\tcc (Cuda coalescente)\n", argv[0]);
		return -1;
	}

	const char * mode = (argc > 1 ? argv[1] : "");
	const int vector_size = (argc > 2 ? atoi(argv[2]) : 0);
	const int block_size = (argc > 3 ? atoi(argv[3]) : 0);
	const int debug = argc > 4 ? strcmp(argv[4], "debug") == 0 : 0;

	srand(time(NULL));

	int state = -1;
	float *x = init(vector_size);
	float *y = init(vector_size);
	float *x_device, *y_device;
	int n_blocks = (vector_size + (block_size - 1)) / block_size;
	int shared_memory = block_size * 2 * sizeof(float);
	float a = 0.0f;

	FILE *f_a;
	FILE *f_x;
	FILE *f_y;
	FILE *f_axy;

	if(argc > 4) {
		a = atof(argv[4]);
	} else {
		a = (float) (rand() % vector_size);
	}

	if(strncmp(mode, "s", 1) == 0) {
		state = SEQUENTIAL;
		if(debug) {
			f_a = fopen("y_s.txt", "w");
			f_x = fopen("x_s.txt", "w");
			f_y = fopen("y_s.txt", "w");
			f_axy = fopen("axy_s.txt", "w");
		}
	} else if(strlen(mode) == 1 && strncmp(mode, "c", 1) == 0) {
		state = CUDA;
		if(debug) {
			f_a = fopen("y_c.txt", "w");
			f_x = fopen("x_c.txt", "w");
			f_y = fopen("y_c.txt", "w");
			f_axy = fopen("axy_c.txt", "w");
		}
	} else if(strncmp(mode, "cc", 2) == 0) {
		state = CUDA_COALESCE;
		if(debug) {
			f_a = fopen("y_c.txt", "w");
			f_x = fopen("x_cc.txt", "w");
			f_y = fopen("y_cc.txt", "w");
			f_axy = fopen("axy_cc.txt", "w");
		}
	}

	if(debug) {
		fprintf(f_a, "%f", a);
		fprint_mat(f_x, x, vector_size);
		fprint_mat(f_y, y, vector_size);
	}

	if(state == SEQUENTIAL) {
		a_x_mais_y_host(a, x, y, vector_size);
	} else {
		alloc_on_device(x, y, (void**)&x_device, (void**) &y_device, vector_size);

		if(state == CUDA) {
			a_x_mais_y_device<<<n_blocks, block_size>>>(a, x_device, y_device, vector_size);
		} else if(state == CUDA_COALESCE) {
			a_x_mais_y_device_coalesce<<<n_blocks, block_size, shared_memory>>>(a, x_device, y_device, vector_size);
		}

		copy_from_device(y_device, y, vector_size);
	}

	if(debug) {
		fprint_mat(f_axy, y, vector_size);
		fclose(f_x);
		fclose(f_y);
		fclose(f_axy);
		fclose(f_a);
	}

	free(x);
	free(y);

	return 0;
}

void fprint_mat(FILE *f, float *v, int n) {
	for(int i = 0; i < n-1; i++) {
		fprintf(f, "%f ", v[i]);
	}

	fprintf(f, "%f", v[n-1]);
}

void alloc_on_device(float *x, float *y, void **x_ptr, void **y_ptr, int n) {
	size_t size = sizeof(float) * n;
	CUDA_CHECK_RETURN(hipMalloc(x_ptr, size));
	CUDA_CHECK_RETURN(hipMalloc(y_ptr, size));
	CUDA_CHECK_RETURN(hipMemcpy(*x_ptr, x, size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(*y_ptr, y, size, hipMemcpyHostToDevice));
}

void copy_from_device(void *v_ptr, float *v, int n) {
	hipMemcpy(v, v_ptr, sizeof(float) * n, hipMemcpyDeviceToHost);
}

void free_on_device(void *x_ptr, void *y_ptr) {
	hipFree(x_ptr);
	hipFree(y_ptr);
}

void a_x_mais_y_host(int a, float *x, float* y, int n) {
	for(int i = 0; i < n; ++i) {
		y[i] = (a * x[i]) + y[i];
	}
}

float* init(int n) {
	float *v = (float*) malloc(sizeof(float) * n);

	for(int i = 0; i < n; ++i) {
		v[i] = (float) i;
	}

	return v;
}

////////////////////////////////////////////////////////////////////////////////

__global__ void a_x_mais_y_device(int a, float* x, float* y, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i < n) {
		y[i] = a * x[i] + y[i];
	}
}

__global__ void a_x_mais_y_device_coalesce(int a, float* x, float* y, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int ti = threadIdx.x;

	extern __shared__ float sharedX[];
	float *sharedY = &sharedX[0] + blockDim.x;

	if(i < n) {
		sharedX[ti] = x[i];
		sharedY[ti] = y[i];

		y[i] = a * sharedX[ti] + sharedY[ti];
	}
}
