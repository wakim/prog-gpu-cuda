/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	for (int i=0; i< count; i++) {
		hipGetDeviceProperties(&prop, i);

		printf( " --- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap: " );

		if (prop.deviceOverlap)
				printf( "Enabled\n" );
		else
				printf( "Disabled\n" );
		printf( "Kernel execution timeout : ");

		if (prop.kernelExecTimeoutEnabled)
				printf( "Enabled\n" );
		else
				printf( "Disabled\n" );

		printf( " --- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( " --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n", prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0],
			prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]
		);
		printf( "Max grid dimensions: (%d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1],
		prop.maxGridSize[2] );
		printf( "\n" );
	}
}
