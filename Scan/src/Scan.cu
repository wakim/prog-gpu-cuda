/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static const int VECTOR_SIZE = 32;
static const int WORK_SIZE = 16;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ int scan_warp(int *ptr, const unsigned int idx = threadIdx.x) {
	const unsigned int aux = idx & 31;

	if(aux >= 1) ptr[idx] = ptr[idx - 1] + ptr[idx];
	if(aux >= 2) ptr[idx] = ptr[idx - 2] + ptr[idx];
	if(aux >= 4) ptr[idx] = ptr[idx - 4] + ptr[idx];
	if(aux >= 8) ptr[idx] = ptr[idx - 8] + ptr[idx];
	if(aux >= 16) ptr[idx] = ptr[idx - 16] + ptr[idx];

	return ptr[idx];
}

__device__ int scan_block(int *ptr, const unsigned int idx = threadIdx.x) {
	const unsigned int aux = idx & 31;
	const unsigned int warp_id = idx >> 5;

	__shared__ int temp[32];

	int val = scan_warp(ptr, idx);

	__syncthreads();

	if(aux == 31) temp[warp_id] = ptr[idx];

	__syncthreads();

	if(warp_id == 0) scan_warp(temp, idx);

	__syncthreads();

	if(warp_id > 0) val = temp[warp_id - 1] + val;

	__syncthreads();

	ptr[idx] = val;

	__syncthreads();

	return val;
}

__global__ void scan_global(int *ptr) {
	const unsigned int idx = threadIdx.x;
	const unsigned int aux = idx & blockDim.x;
	const unsigned int bi = blockIdx.x;

	extern __shared__ int temp[];

	int val = scan_block(ptr, idx);

	__syncthreads();

	if(aux == blockDim.x) temp[bi] = ptr[idx];

	__syncthreads();

	if(bi == 0) scan_block(temp, idx);

	__syncthreads();

	if(bi > 0) val = temp[bi - 1] + val;

	__syncthreads();

	ptr[idx] = val;

	__syncthreads();

	return;
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	int *d = NULL;
	int i;
	unsigned int GRID_SIZE = (1 + VECTOR_SIZE) / WORK_SIZE;
	unsigned int idata[VECTOR_SIZE], odata[VECTOR_SIZE];

	for (i = 0; i < VECTOR_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * VECTOR_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * VECTOR_SIZE, hipMemcpyHostToDevice));

	scan_global<<<GRID_SIZE, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * VECTOR_SIZE, hipMemcpyDeviceToHost));

	unsigned int sum = idata[0];
	for (i = 0; i < VECTOR_SIZE - 1; i++) {
		if(odata[i] != sum) {
			printf("Erro !!!");
		}
		if((i + 1) < VECTOR_SIZE) {
			sum += idata[i+1];
		}
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);
	}

	if(odata[VECTOR_SIZE - 1] != sum) {
		printf("Erro !!!");
	}
	printf("Input value: %u, device output: %u\n", idata[i], odata[VECTOR_SIZE - 1]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
