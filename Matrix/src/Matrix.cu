// Programação Paralela e Distribuída (Programação paralela em GPUs) (MAB622) (DCC/UFRJ)
// Outubro de 2012
// Prof.: Silvana Rossetto
// Laboratório 1: Introducao ao ambiente de programação CUDA no Linux

// Multiplica duas float*es: A * B = C


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cstring>

// Thread block size
const int BLOCK_SIZE = 16;
const int SHARED_SIZE = 16;
const int WA = 1024; // Matrix A width
const int HA = 1024; // Matrix A height
const int WB = HA;// Matrix B width
const int HB = WA; // Matrix B height
const int WC = WA; // Matrix C width
const int HC = HB; // Matrix C height

const int CUDA_COALESC = 1;
const int CUDA = 2;

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

////////////////////////// CUDA Kernel /////////////////////////
__global__ void
matrix_mult_coalesc(float* C, float* A, float* B, int width, int height) {
	__shared__ float As[SHARED_SIZE][SHARED_SIZE];
	__shared__ float Bs[SHARED_SIZE][SHARED_SIZE];

	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	int row = by * BLOCK_SIZE + ty;
	int col = bx * BLOCK_SIZE + tx;
	int gridDimX = width / BLOCK_SIZE;

	float elementC = 0.0;
	for(int m = 0; m < gridDimX; ++m) {
		if(row < height && col < width) {
			As[ty][tx] = A[(row * width) + (m * BLOCK_SIZE + tx)];
			Bs[ty][tx] = B[(m * BLOCK_SIZE + ty) * width + col];
		}
		__syncthreads();
		if(row < height && col < width) {
			for(int k = 0; k < BLOCK_SIZE; ++k) {
				elementC += As[ty][k] * Bs[k][tx];
			}
		}
		__syncthreads();
	}

	// Write the matrix to device memory each
	// thread writes one element
	C[row * width + col] = elementC;
}

__global__ void
matrix_mult(float *c, float *a, float *b, int width, int height) {
	int line = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	float c_element = 0;

	if(line >= height || column >= width) {
		return;
	}

	for(int k = 0; k < width; ++k) {
		c_element += a[line * width + k] * b[k * width + column];
	}

	c[line * width + column] = c_element;
}

// Inicializa uma float* de float (inteiros consecutivos)
void initMat(float* data, size_t size, int mod) {
	int _mod = 13 + ((mod % 2 == 0) ? (mod + 1) : mod);

	for (int i = 0; i < size; ++i) {
		data[i] = (float)(i % _mod) + 1;
	}
}

void fprintMat(FILE* input, float* data, size_t size, int w) {
	for(int i = 0; i < size; ++i) {
		fprintf(input, "%f ", data[i]);
		if(((i + 1) % w) == 0) {
			fprintf(input, "\n");
		}
	}
}

// Imprime uma float* de float
void printMat(float* data, size_t size, int w) {
	fprintMat(stdin, data, size, w);
}

float* fscanMat(FILE* input, size_t size) {
	float* m = (float*) malloc(sizeof(float) * size);

	for(int i = 0; i < size; ++i) {
		fscanf(input, "%f", m+i);
	}

	return m;
}

float* scanMat(size_t size) {
	return fscanMat(stdin, size);
}

//funcao main
int main(int argc, char** argv) {
	if(argc < 2) {
		printf("Usage: %s <algorithm> <debug opcional>\nalgorithm:\n\t c (Cuda)\n\t cc (Cuda Coalesc)\n", argv[0]);
		return -1;
	}

	const int debug = (argc > 2 ? strcmp("debug", argv[2]) == 0 : 0);

	int state = -1;
	//1. aloca memoria para as float*es de entrada A e B no host
	size_t size_A = WA * HA;
	size_t mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*) malloc(mem_size_A);

	size_t size_B = WB * HB;
	size_t mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*) malloc(mem_size_B);

	FILE* f_A = NULL;
	FILE* f_B = NULL;
	FILE* f_C = NULL;

	if(strlen(argv[1]) == 1 && strncmp("c", argv[1], 1) == 0) {
		state = CUDA;
		if(debug) {
			f_A = fopen("m_A_c.txt", "w");
			f_B = fopen("m_B_c.txt", "w");
			f_C = fopen("m_C_c.txt", "w");
		}
	} else if(strncmp("cc", argv[1], 2) == 0) {
		state = CUDA_COALESC;
		if(debug) {
			f_A = fopen("m_A_cc.txt", "w");
			f_B = fopen("m_B_cc.txt", "w");
			f_C = fopen("m_C_cc.txt", "w");
		}
	}

	//2. inicializa as float'es de entrada
	initMat(h_A, size_A, WA);

	initMat(h_B, size_B, WB);

	//3. (opcional) imprime as float*es de entrada
	if(debug) {
		fprintMat(f_A, h_A, size_A, WA);
		fprintMat(f_B, h_B, size_B, WB);
	}

	//4. aloca memoria para a float* de saida C no host
	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*) malloc(mem_size_C);

	//5. aloca memoria no device para as tres float*es (A, B e C)
	float* d_A;
	float* d_B;
	float* d_C;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d_A, mem_size_A));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_B, mem_size_B));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_C, mem_size_C));

	//6. copia as float*es de entrada do host para o device
	CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

	//7. executa a multiplicacao
	// seta os parametros de configuracao do kernel
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(WC/threads.x, HC/threads.y);

	grid.x = (grid.x == 0 ? 1 : grid.x);
	grid.y = (grid.y == 0 ? 1 : grid.y);

	// executa o kernel
	if(state == CUDA) {
		matrix_mult<<<grid, threads>>>(d_C, d_A, d_B, WC, HC);
	} else {
		matrix_mult_coalesc<<<grid, threads>>>(d_C, d_A, d_B, WC, HC);
	}

	CUDA_CHECK_RETURN(hipGetLastError());

	//8. copia os resultados do device para o host
	CUDA_CHECK_RETURN(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

	//9. imprime a float* de saida
	if(debug) {
		fprintMat(f_C, h_C, size_C, WC);
		fclose(f_A);
		fclose(f_B);
		fclose(f_C);
	}

	//10. libera os espacos de memoria
	free(h_A);
	free(h_B);
	free(h_C);

	CUDA_CHECK_RETURN(hipFree(d_A));
	CUDA_CHECK_RETURN(hipFree(d_B));
	CUDA_CHECK_RETURN(hipFree(d_C));
}
